#include "hip/hip_runtime.h"
/** @author Claudia Pfreundt */

#define CHECK_ERRORS
//#define PROFILING_ACTIVE
//#define DEBUG_ON

#define VECTOR_DIM 3
#define MATRIX_DIM 9

#include <dbot/models/observation_models/kinect_image_observation_model_gpu/cuda_filter.hpp>
#include <GL/glut.h>


#include <stdio.h>
#include <stdlib.h>
#include <iostream>


#include <hip/hip_runtime.h>
#include "cuda_gl_interop.h"
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <hip/hip_math_constants.h>


using namespace std;

namespace fil
{

// ====================== CUDA CONSTANT VALUES ======================= //

// used in propagate
// 1000 denotes the maximum number of objects
__constant__ float3 g_rot_center[1000];
// sigmas.x == angle_sigma, sigmas.y == trans_sigma
__constant__ float2 g_sigmas;

// used in propagateOcclusion
__constant__ float g_p_visible_occluded;
__constant__ float g_c;
__constant__ float g_log_c;

// used in prob
__constant__ float g_tail_weight;
__constant__ float g_model_sigma;
__constant__ float g_sigma_factor;
__constant__ float g_max_depth;
__constant__ float g_exponential_rate;

// used in compare
__constant__ float g_p_visible_init;

// texture for OpenGL interop
texture<float, hipTextureType2D, hipReadModeElementType> texture_reference;





// ************************************************************************************** //
// ************************************************************************************** //
// ================================== CUDA KERNELS ====================================== //
// ************************************************************************************** //
// ************************************************************************************** //

// ============================================================================================= //
// ====================== DEVICE kernels - to be called by other kernels ======================= //
// ============================================================================================= //

// ====================== MATRIX MANIPULATION FUNCTIONS ======================= //

__device__ void multiply_matrices(float *A, float *B, float *C) {
    float sum = 0;
    for (int i = 0; i < VECTOR_DIM; i++) {        // iterate through rows
        for (int j = 0; j < VECTOR_DIM; j++) {    // iterate through cols
            for (int k = 0; k < VECTOR_DIM; k++) {
                sum += A[i * VECTOR_DIM + k] * B[k * VECTOR_DIM + j];
            }
            C[i * VECTOR_DIM + j] = sum;
            sum = 0;
        }
    }
}

__device__ float3 multiply_matrix_with_vector(float* M, float3 v) {
    float result[3];
    float v_copy[3];
    v_copy[0] = v.x; v_copy[1] = v.y; v_copy[2] = v.z;
    float sum = 0;

    for (int i = 0; i < VECTOR_DIM; i++) {
        for (int j = 0; j < VECTOR_DIM; j++) {
            sum += M[i * VECTOR_DIM + j] * v_copy[j];
        }
        result[i] = sum;
        sum = 0;
    }

    return make_float3(result[0], result[1], result[2]);
}

/* axis is defined as follows: 0 = x, 1 = y, 2 = z */
__device__ void create_rotation_matrix(const float angle, const int axis, float *R) {
    float cos_angle = cos(angle);
    float sin_angle = sin(angle);

    if (axis == 0) {
        R[0] = 1;   R[1] = 0;           R[2] = 0;
        R[3] = 0;   R[4] = cos_angle;   R[5] = -sin_angle;
        R[6] = 0;   R[7] = sin_angle;   R[8] = cos_angle;
    } else if (axis == 1) {
        R[0] = cos_angle;   R[1] = 0;   R[2] = sin_angle;
        R[3] = 0;           R[4] = 1;   R[5] = 0;
        R[6] = -sin_angle;  R[7] = 0;   R[8] = cos_angle;
    } else if (axis == 2) {
        R[0] = cos_angle;   R[1] = -sin_angle;  R[2] = 0;
        R[3] = sin_angle;   R[4] = cos_angle;   R[5] = 0;
        R[6] = 0;           R[7] = 0;           R[8] = 1;
    }
}

__device__ void transpose_matrix(float *A, float *T) {
    T[0] = A[0];
    T[1] = A[3];
    T[2] = A[6];
    T[3] = A[1];
    T[4] = A[4];
    T[5] = A[7];
    T[6] = A[2];
    T[7] = A[5];
    T[8] = A[8];
}

// ====================== VECTOR MANIPULATION FUNCTIONS ======================= //

__device__ float4 normalize(const float4 v) {
    float4 v_n = v;
    const float n = 1.0f/sqrt(v_n.x*v_n.x+v_n.y*v_n.y+v_n.z*v_n.z+v_n.w*v_n.w);
    v_n.x *= n;
    v_n.y *= n;
    v_n.z *= n;
    v_n.w *= n;

    return v_n;
}

__device__ float3 operator+(const float3 &a, const float3 &b) {

  return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);

}

__device__ float3 negate(const float3 &a) {
    return make_float3(-a.x, -a.y, -a.z);
}

// ======================= QUATERNION CONVERSIONS AND MANIPULATION FUNCTIONS ======================= //

__device__ void quaternion_to_matrix(const float4 q_in, float *Q) {
    float4 q = normalize(q_in);
    Q[0] = 1.0f - 2.0f*q.y*q.y - 2.0f*q.z*q.z;  Q[1] = 2.0f*q.x*q.y - 2.0f*q.z*q.w;         Q[2] = 2.0f*q.x*q.z + 2.0f*q.y*q.w;
    Q[3] = 2.0f*q.x*q.y + 2.0f*q.z*q.w;         Q[4] = 1.0f - 2.0f*q.x*q.x - 2.0f*q.z*q.z;  Q[5] = 2.0f*q.y*q.z - 2.0f*q.x*q.w;
    Q[6] = 2.0f*q.x*q.z - 2.0f*q.y*q.w;         Q[7] = 2.0f*q.y*q.z + 2.0f*q.x*q.w;         Q[8] = 1.0f - 2.0f*q.x*q.x - 2.0f*q.y*q.y;
}

__device__ float4 matrix_to_quaternion(float *Q) {
    float4 q;

    q.w = sqrtf( fmaxf( 0, 1 + Q[0] + Q[4] + Q[8] ) ) / 2;
    q.x = sqrtf( fmaxf( 0, 1 + Q[0] - Q[4] - Q[8] ) ) / 2;
    q.y = sqrtf( fmaxf( 0, 1 - Q[0] + Q[4] - Q[8] ) ) / 2;
    q.z = sqrtf( fmaxf( 0, 1 - Q[0] - Q[4] + Q[8] ) ) / 2;
    if (( q.x * ( Q[7] - Q[5] ) ) < 0) {
        q.x = -q.x;
    }
    if (( q.y * ( Q[2] - Q[6] ) ) < 0) {
        q.y = -q.y;
    }
    if (( q.z * ( Q[3] - Q[1] ) ) < 0) {
        q.z = -q.z;
    }

    return q;
}

__device__ float4 multiply_quaternions(float4 q1, float4 q2) {
    float w = (q1.w * q2.w) - (q1.x * q2.x) - (q1.y * q2.y) - (q1.z * q2.z);
    float x = (q1.w * q2.x) + (q1.x * q2.w) + (q1.y * q2.z) - (q1.z * q2.y);
    float y = (q1.w * q2.y) - (q1.x * q2.z) + (q1.y * q2.w) + (q1.z * q2.x);
    float z = (q1.w * q2.z) + (q1.x * q2.y) - (q1.y * q2.x) + (q1.z * q2.w);

    return make_float4(x, y, z, w);
}



// ======================= helper functions for compare (observation model)  ======================= //

__device__ float propagate_occlusion(float initial_p_source, float time) {
    if (isnan(time)) {
        return initial_p_source;
    }
    float pow_c_time = exp(time*g_log_c);
    return (float) pow_c_time*initial_p_source + g_p_visible_occluded*(pow_c_time-1.)/(g_c-1.);
}



__device__ float prob(float observation, float prediction, bool visible)
{
    // todo: if the prediction is infinite, the prob should not depend on visibility. it does not matter
    // for the algorithm right now, but it should be changed

    float sigma = g_model_sigma + g_sigma_factor*observation*observation;
    if(visible)
    {
        if(isinf(prediction)) // if the prediction is infinite we return the limit
            return g_tail_weight/g_max_depth;
        else
            return g_tail_weight/g_max_depth
                    + (1 - g_tail_weight)*expf(-(powf(prediction-observation,2)/(2*sigma*sigma)))
                    / (sqrtf(2*M_PI) *sigma);
    }
    else
    {
        if(isinf(prediction)) // if the prediction is infinite we return the limit
            return g_tail_weight/g_max_depth +
                    (1-g_tail_weight)*g_exponential_rate*
                    expf(0.5*g_exponential_rate*(-2*observation + g_exponential_rate*sigma*sigma));

        else
            return g_tail_weight/g_max_depth +
                    (1-g_tail_weight)*g_exponential_rate*
                    expf(0.5*g_exponential_rate*(2*prediction-2*observation + g_exponential_rate*sigma*sigma))
        *(1+erff((prediction-observation+g_exponential_rate*sigma*sigma)/(sqrtf(2)*sigma)))
        /(2*(expf(prediction*g_exponential_rate)-1));
    }
}






// ============================================================================================= //
// ========================= GLOBAL kernels - to be called by CPU code ========================= //
// ============================================================================================= //



__global__ void setup_number_generators_kernel(int current_time, hiprandStateMRG32k3a *mrg_state, int n_poses)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n_poses) {
        /* Each thread gets same seed, a different sequence number, no offset */
        hiprand_init(current_time, id, 0, &mrg_state[id]);
    }
}


__global__ void propagate_kernel(float *states, int n_states, int states_size, float delta_time, hiprandStateMRG32k3a *mrg_state)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n_states) {

        /* Copy sigmas from constant memory into local register */
        float2 local_sigmas = g_sigmas;

        /* Copy mrg_state from global memory into local register */
        hiprandStateMRG32k3a local_mrg_state = mrg_state[id];        

        for (int i = 0; i < states_size / 7; i++) {

            int states_index = id * states_size + i * 7;

            /* Copy rot_center from constant memory into local register */
            float3 local_rot_center = g_rot_center[i];

            /* TODO coalesce accesses? Does it do it automatically or do I manually have to store them as
            * float3 and float4 values? */
            /* quaternion stored as (w,x,y,z), but make_float4 takes (x,y,z,w) */
            float4 q_init_vector = make_float4(states[states_index + 1], states[states_index + 2], states[states_index + 3], states[states_index]);
            float3 t_init = make_float3(states[states_index + 4], states[states_index + 5], states[states_index + 6]);

            float angle_x, angle_y, angle_z;
            float trans_x, trans_y, trans_z;

            angle_x = hiprand_normal(&local_mrg_state) * delta_time * local_sigmas.x;
            angle_y = hiprand_normal(&local_mrg_state) * delta_time * local_sigmas.x;
            angle_z = hiprand_normal(&local_mrg_state) * delta_time * local_sigmas.x;
            trans_x = hiprand_normal(&local_mrg_state) * delta_time * local_sigmas.y;
            trans_y = hiprand_normal(&local_mrg_state) * delta_time * local_sigmas.y;
            trans_z = hiprand_normal(&local_mrg_state) * delta_time * local_sigmas.y;


            float q_rand_matrix[MATRIX_DIM];
            float q_init_matrix[MATRIX_DIM];

            float rot_matrix_x[MATRIX_DIM];
            float rot_matrix_y[MATRIX_DIM];
            float rot_matrix_z[MATRIX_DIM];

            float tmp_matrix[MATRIX_DIM];


            float3 t_rand = make_float3(trans_x, trans_y, trans_z);

            create_rotation_matrix(angle_x, 0, rot_matrix_x);
            create_rotation_matrix(angle_y, 1, rot_matrix_y);
            create_rotation_matrix(angle_z, 2, rot_matrix_z);

            multiply_matrices(rot_matrix_y, rot_matrix_z, tmp_matrix);
            multiply_matrices(rot_matrix_x, tmp_matrix, q_rand_matrix);

            float4 q_rand_vector = matrix_to_quaternion(q_rand_matrix);

            quaternion_to_matrix(q_init_vector, q_init_matrix);

            float3 t = negate(multiply_matrix_with_vector(q_init_matrix, multiply_matrix_with_vector(q_rand_matrix, local_rot_center)))
                   + multiply_matrix_with_vector(q_init_matrix, local_rot_center)
                   + t_init
                   + t_rand;

            float4 q = multiply_quaternions(q_init_vector, q_rand_vector);
            q = normalize(q);

            /* write state back into global memory */
            states[states_index] = q.w;
            states[states_index + 1] = q.x;
            states[states_index + 2] = q.y;
            states[states_index + 3] = q.z;

            states[states_index + 4] = t.x;
            states[states_index + 5] = t.y;
            states[states_index + 6] = t.z;
        }

        /* Copy mrg state back to global memory */
        mrg_state[id] = local_mrg_state;
    }
}






__global__ void propagate_multiple_kernel(float *states, int n_states, int n_objects, int states_size, float delta_time, hiprandStateMRG32k3a *mrg_state)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n_states) {

        /* Copy sigmas from constant memory into local register */
        float2 local_sigmas = g_sigmas;

        /* Copy mrg_state from global memory into local register */
        hiprandStateMRG32k3a local_mrg_state = mrg_state[id];

        for (int i = 0; i < n_objects; i++) {

            int states_index = id * n_objects * states_size + i * states_size;

            /* Copy rot_center from constant memory into local register */
            float3 local_rot_center = g_rot_center[i];

            /* TODO coalesce accesses? Does it do it automatically or do I manually have to store them as
            * float3 and float4 values? */
            /* quaternion stored as (w,x,y,z), but make_float4 takes (x,y,z,w) */
            float4 q_init_vector = make_float4(states[states_index + 1], states[states_index + 2], states[states_index + 3], states[states_index]);
            float3 t_init = make_float3(states[states_index + 4], states[states_index + 5], states[states_index + 6]);

            float angle_x, angle_y, angle_z;
            float trans_x, trans_y, trans_z;

            // WARNING: same random number for all states, because mrg_state is the same..
            angle_x = hiprand_normal(&local_mrg_state) * delta_time * local_sigmas.x;
            angle_y = hiprand_normal(&local_mrg_state) * delta_time * local_sigmas.x;
            angle_z = hiprand_normal(&local_mrg_state) * delta_time * local_sigmas.x;
            trans_x = hiprand_normal(&local_mrg_state) * delta_time * local_sigmas.y;
            trans_y = hiprand_normal(&local_mrg_state) * delta_time * local_sigmas.y;
            trans_z = hiprand_normal(&local_mrg_state) * delta_time * local_sigmas.y;


            float q_rand_matrix[MATRIX_DIM];
            float q_init_matrix[MATRIX_DIM];

            float rot_matrix_x[MATRIX_DIM];
            float rot_matrix_y[MATRIX_DIM];
            float rot_matrix_z[MATRIX_DIM];

            float tmp_matrix[MATRIX_DIM];


            float3 t_rand = make_float3(trans_x, trans_y, trans_z);

            create_rotation_matrix(angle_x, 0, rot_matrix_x);
            create_rotation_matrix(angle_y, 1, rot_matrix_y);
            create_rotation_matrix(angle_z, 2, rot_matrix_z);

            multiply_matrices(rot_matrix_y, rot_matrix_z, tmp_matrix);
            multiply_matrices(rot_matrix_x, tmp_matrix, q_rand_matrix);

            float4 q_rand_vector = matrix_to_quaternion(q_rand_matrix);

            quaternion_to_matrix(q_init_vector, q_init_matrix);

            float3 t = negate(multiply_matrix_with_vector(q_init_matrix, multiply_matrix_with_vector(q_rand_matrix, local_rot_center)))
                   + multiply_matrix_with_vector(q_init_matrix, local_rot_center)
                   + t_init
                   + t_rand;

            float4 q = multiply_quaternions(q_init_vector, q_rand_vector);
            q = normalize(q);

            /* write state back into global memory */
            states[states_index] = q.w;
            states[states_index + 1] = q.x;
            states[states_index + 2] = q.y;
            states[states_index + 3] = q.z;

            states[states_index + 4] = t.x;
            states[states_index + 5] = t.y;
            states[states_index + 6] = t.z;
        }

        /* Copy mrg state back to global memory */
        mrg_state[id] = local_mrg_state;
    }
}












__global__ void compare_kernel(float *observations, float* visibility_probs, int n_pixels_per_pose,
                        bool constant_occlusion, float *d_log_likelihoods, float delta_time, int n_poses, int n_rows, int n_cols) {
    int block_id = blockIdx.x + blockIdx.y * gridDim.x;
    if (block_id < n_poses) {

        int pixel_nr = threadIdx.x;
//        int pixel_nr = threadIdx.x * ceilf(n_pixels_per_pose / blockDim.x);
        int global_index = block_id * n_pixels_per_pose + pixel_nr;

        // OpenGL contructs the texture so that the left lower edge is (0,0), but our observations texture
        // has its (0,0) in the upper left corner, so we need to reverse the reads from the OpenGL texture.
        float texture_array_index_x = blockIdx.x * n_cols + pixel_nr % n_cols;
        float texture_array_index_y = gridDim.y * n_rows - (blockIdx.y * n_rows + pixel_nr / n_cols + 1);

        float depth;
        float observed_depth;
        float visibility_prob = g_p_visible_init;
        float local_sum_of_likelihoods = 0;
        float p_obsIpred_vis, p_obsIpred_occl, p_obsIinf;

        // TODO: uninitialized?
        __shared__ float log_likelihoods;

        if (threadIdx.x == 0) {
            log_likelihoods = 0;
        }

        __syncthreads();

        while (pixel_nr < n_pixels_per_pose ) { //&& pixel_nr < (threadIdx.x + 1) * ceilf(n_pixels_per_pose / blockDim.x)) {

            depth = tex2D(texture_reference, texture_array_index_x, texture_array_index_y);
            observed_depth = observations[pixel_nr];

            // TODO either this, or only write the values back for pixels with depth value == 1.
            // Could save some data transfer time, but will cost more execution time, since all
            // the threads in one warp will have to wait for the else-branch to finish
            if (!constant_occlusion) {
                visibility_prob = propagate_occlusion(visibility_probs[global_index], delta_time);
                visibility_probs[global_index] = visibility_prob;
            }
//            if (!constant_occlusion) {
//                visibility_prob = propagateOcclusion(visibility_probs[global_index], delta_time);
//            }

            //TODO slow: 4800 threads have to go through this whole if instruction
            if (depth != 0 && !isnan(observed_depth)) {

                // prob of observation given prediction, knowing that the object is visible
                p_obsIpred_vis = prob(observed_depth, depth, true) * visibility_prob;
                // prob of observation given prediction, knowing that the object is occluded
                p_obsIpred_occl = prob(observed_depth, depth, false) * (1-visibility_prob);
                // prob of observation given no intersection
                p_obsIinf = prob(observed_depth, HIP_INF_F, false);

                local_sum_of_likelihoods += logf((p_obsIpred_vis + p_obsIpred_occl)/p_obsIinf);

                if(!constant_occlusion) { // we check if we are tracking the visibilities
                    // we update the visibility (occlusion) with the observations
                    visibility_probs[global_index] = p_obsIpred_vis/(p_obsIpred_vis + p_obsIpred_occl);
                }
//                if (!constant_occlusion) {
//                    visibility_prob = p_obsIpred_vis/(p_obsIpred_vis + p_obsIpred_occl);
//                }
            }

//            if (!constant_occlusion) {
//                visibility_probs[global_index] = visibility_prob;
//            }

            pixel_nr += blockDim.x;
//            pixel_nr += 1;
            global_index = block_id * n_pixels_per_pose + pixel_nr;
            texture_array_index_x = blockIdx.x * n_cols + pixel_nr % n_cols;
            texture_array_index_y = gridDim.y * n_rows - (blockIdx.y * n_rows + pixel_nr / n_cols + 1);
        }

        // TODO: will execute blockDim.x sequential writes to log_likelihoods
        // instead could do a manual reduction after syncthreads
        atomicAdd(&log_likelihoods, local_sum_of_likelihoods);

        __syncthreads();

        if (threadIdx.x == 0) {
            d_log_likelihoods[block_id] = log_likelihoods;
        }
    } else {
        __syncthreads();
    }

}





__global__ void compare_multiple_kernel(float *observations, float* old_visibility_probs, float* new_visibility_probs, int* occlusion_image_indices, int nr_pixels,
                                 float *d_log_likelihoods, float delta_time, int n_poses, int n_rows, int n_cols, bool update) {
    int block_id = blockIdx.x + blockIdx.y * gridDim.x;
    if (block_id < n_poses) {

        int pixel_nr = threadIdx.x;

        // OpenGL contructs the texture so that the left lower edge is (0,0), but our observations texture
        // has its (0,0) in the upper left corner, so we need to reverse the reads from the OpenGL texture.
        float texture_array_index_x = blockIdx.x * n_cols + pixel_nr % n_cols;

        float texture_array_index_y = gridDim.y * n_rows - 1 - (blockIdx.y * n_rows + pixel_nr / n_cols);

        float depth;
        float observed_depth;
        float visibility_prob = g_p_visible_init;
        float local_sum_of_likelihoods = 0;
        float p_obsIpred_vis, p_obsIpred_occl, p_obsIinf;

        __shared__ float log_likelihoods;
        __shared__ int occlusion_image_index;

        if (threadIdx.x == 0) {
            log_likelihoods = 0;
            occlusion_image_index = occlusion_image_indices[block_id];
        }

        __syncthreads();

        float* visibility_probs = old_visibility_probs;
        int occlusion_pixel_index= occlusion_image_index * nr_pixels + pixel_nr;

        if (update) {
            // copy / duplicate visibility probabilities from the old particles
            int index_from_visibility = occlusion_image_indices[block_id] * nr_pixels;
            int index_to_visibility = block_id * nr_pixels;

            while (pixel_nr < nr_pixels) {
                new_visibility_probs[index_to_visibility + pixel_nr] = old_visibility_probs[index_from_visibility + pixel_nr];
                pixel_nr += blockDim.x;
            }

            // change visibility prob array to the new one and change the global index
            visibility_probs = new_visibility_probs;
            // reset pixel_nr for following loop
            pixel_nr = threadIdx.x;

            occlusion_pixel_index= block_id * nr_pixels + pixel_nr;
        }


        while (pixel_nr < nr_pixels ) {

            depth = tex2D(texture_reference, texture_array_index_x, texture_array_index_y);
            observed_depth = observations[pixel_nr];

            // TODO either this, or only write the values back for pixels with depth value == 1.
            // Could save some data transfer time, but will cost more execution time, since all
            // the threads in one warp will have to wait for the else-branch to finish

            visibility_prob = propagate_occlusion(visibility_probs[occlusion_pixel_index], delta_time);
            if (update) visibility_probs[occlusion_pixel_index] = visibility_prob;


            if (depth != 0 && !isnan(observed_depth)) {


                // prob of observation given prediction, knowing that the object is visible
                p_obsIpred_vis = prob(observed_depth, depth, true) * visibility_prob;
                // prob of observation given prediction, knowing that the object is occluded
                p_obsIpred_occl = prob(observed_depth, depth, false) * (1-visibility_prob);
                // prob of observation given no intersection
                p_obsIinf = prob(observed_depth, HIP_INF_F, false);

                local_sum_of_likelihoods += logf((p_obsIpred_vis + p_obsIpred_occl)/p_obsIinf);


                if(update) {
                    // we update the visibility (occlusion) probability with the observations
                    visibility_probs[occlusion_pixel_index] = p_obsIpred_vis/(p_obsIpred_vis + p_obsIpred_occl);
                }
            }

            pixel_nr += blockDim.x;
            occlusion_pixel_index += blockDim.x;
            texture_array_index_x = blockIdx.x * n_cols + pixel_nr % n_cols;
            texture_array_index_y = gridDim.y * n_rows - (blockIdx.y * n_rows + pixel_nr / n_cols + 1);
        }

        // TODO: will execute blockDim.x sequential writes to log_likelihoods
        // instead could do a manual reduction after syncthreads
        atomicAdd(&log_likelihoods, local_sum_of_likelihoods);

        __syncthreads();

        if (threadIdx.x == 0) {
            d_log_likelihoods[block_id] = log_likelihoods;
        }
    } else {
        __syncthreads();
    }

}







__global__ void resample_kernel(float *visibility_probs,
                         float *visibility_probs_copy,
                         float *states,
                         float *states_copy,
                         int *resampling_indices,
                         int nr_pixels,
                         int nr_features) {

    int pixel_nr = threadIdx.x;
    int feature_nr = threadIdx.x;
    int index_from_visibility = resampling_indices[blockIdx.x] * nr_pixels;
    int index_to_visibility = blockIdx.x * nr_pixels;
    int index_from_states = resampling_indices[blockIdx.x] * nr_features;
    int index_to_states = blockIdx.x * nr_features;


    while (pixel_nr < nr_pixels) {
        visibility_probs_copy[index_to_visibility + pixel_nr] = visibility_probs[index_from_visibility + pixel_nr];
        pixel_nr += blockDim.x;
    }
    while (feature_nr < nr_features) {
        states_copy[index_to_states + feature_nr] = states[index_from_states + feature_nr];
        feature_nr += blockDim.x;
    }
}


__global__ void resample_multiple_kernel(float *visibility_probs,
                                  float *visibility_probs_copy,
                                  int *resampling_indices,
                                  int nr_pixels) {

    int pixel_nr = threadIdx.x;
    int index_from_visibility = resampling_indices[blockIdx.x] * nr_pixels;
    int index_to_visibility = blockIdx.x * nr_pixels;

    while (pixel_nr < nr_pixels) {
        visibility_probs_copy[index_to_visibility + pixel_nr] = visibility_probs[index_from_visibility + pixel_nr];
        pixel_nr += blockDim.x;
    }
}



// ************************************************************************************** //
// ************************************************************************************** //
// ========================== CUDA_FILTER MEMBER FUNCTIONS ============================== //
// ************************************************************************************** //
// ************************************************************************************** //


CudaFilter::CudaFilter() :
    n_cols_(WINDOW_WIDTH),
    n_rows_(WINDOW_HEIGHT),
    n_poses_set_(false)
{

    hipDeviceProp_t  props;
    int device_number;

    memset( &props, 0, sizeof( hipDeviceProp_t ) );
    props.major = 2;
    props.minor = 0;
    hipChooseDevice( &device_number, &props );
    check_cuda_error("No device with compute capability > 2.0 found");

    /* tell CUDA which device we will be using for graphic interop
     * requires that the CUDA device be specified by
     * cudaGLSetGLDevice() before any other runtime calls. */

    cudaGLSetGLDevice( device_number );
    check_cuda_error("cudaGLsetGLDevice");

    hipGetDeviceProperties(&props, device_number);     // we will run the program only on one graphics card, the first one we can find = 0
    warp_size_ = props.warpSize;            // equals 32 for all current graphics cards, but might change in the future
    n_mps_ = props.multiProcessorCount;

    cuda_device_properties_ = props;

    cout << "Your device has the following properties: " << endl
         << "CUDA Version: " << props.major << "." << props.minor << endl
         << "Number of multiprocessors: " << n_mps_ << endl
         << "Warp size: " << warp_size_ << endl;

    /* each multiprocessor has various KB of memory (64 KB for the GTX 560 Ti 448) which can be subdivided
     * into L1 cache or shared memory. If you don't need a lot of shared memory set this to prefer L1. */
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);


    d_states_ = NULL;
    d_states_copy_ = NULL;
    d_visibility_probs_ = NULL;
    d_visibility_probs_copy_ = NULL;
    d_observations_ = NULL;
    d_log_likelihoods_ = NULL;
    d_mrg_states_ = NULL;
    d_resampling_indices_ = NULL;
    d_prev_sample_indices_ = NULL;

}

void CudaFilter::init(vector<vector<float> > com_models, float angle_sigma, float trans_sigma,
                      float p_visible_init, float c, float log_c, float p_visible_occluded,
                      float tail_weight, float model_sigma, float sigma_factor, float max_depth, float exponential_rate) {

    occlusion_time_ = 0;
    last_propagation_time_ = 0;
    count_ = 0;
    compare_kernel_time_ = 0;
    copy_likelihoods_time_ = 0;
    visibility_prob_default_ = p_visible_init;

    float2 local_sigmas = make_float2(angle_sigma, trans_sigma);

    allocate(d_observations_, n_cols_ * n_rows_ * sizeof(float), "d_observations");
//    allocate(d_log_likelihoods_, sizeof(float) * n_poses_, "d_log_likelihoods");
    // TODO don't allocate here!! only when setting resolution!
//    allocate(d_prev_sample_indices_, sizeof(int) * n_poses_, "d_prev_sample_indices");
//    allocate(d_resampling_indices_, sizeof(int) * n_poses_, "d_resampling_indices");

    hipMemset(d_log_likelihoods_, 0, sizeof(float) * nr_poses_);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemset d_log_likelihoods");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_sigmas), &local_sigmas, sizeof(float2), 0, hipMemcpyHostToDevice);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpyToSymbol local_sigmas -> sigmas");
    #endif

    vector<float3> com_models_raw;
    for (int i = 0; i < com_models.size(); i++) {
        com_models_raw.push_back(make_float3(com_models[i][0], com_models[i][1], com_models[i][2]));
    }

    hipMemcpyToSymbol(HIP_SYMBOL(g_rot_center), com_models_raw.data(), com_models_raw.size() * sizeof(float3), 0, hipMemcpyHostToDevice);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpyToSymbol com_model -> rot_center");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_p_visible_init), &p_visible_init, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpyToSymbol p_visible_init -> g_p_visible_init");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_c), &c, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpyToSymbol c -> g_c");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_log_c), &log_c, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpyToSymbol log_c -> g_log_c");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_p_visible_occluded), &p_visible_occluded, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpyToSymbol p_visible_occluded -> g_p_visible_occluded");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_tail_weight), &tail_weight, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpyToSymbol tail_weight -> g_tail_weight");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_model_sigma), &model_sigma, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpyToSymbol model_sigma -> g_model_sigma");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_sigma_factor), &sigma_factor, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpyToSymbol sigma_factor -> g_sigma_factor");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_max_depth), &max_depth, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpyToSymbol max_depth -> g_max_depth");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_exponential_rate), &exponential_rate, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpyToSymbol exponential_rate -> g_exponential_rate");
    #endif
}


void CudaFilter::propagate(const float &current_time, vector<vector<float> > &states)
{


    float delta_time = current_time - last_propagation_time_;
    last_propagation_time_ = current_time;


    propagate_kernel <<< n_blocks_, nr_threads_ >>> (d_states_, nr_poses_, n_features_, delta_time, d_mrg_states_);
    #ifdef CHECK_ERRORS
        check_cuda_error("propagate kernel call");
    #endif



    // TODO necessary? Remove for performance?
    hipDeviceSynchronize();
    #ifdef CHECK_ERRORS
        check_cuda_error("hipDeviceSynchronize propagate");
    #endif



    float *states_raw = (float*) malloc(nr_poses_ * n_features_ * sizeof(float));
    hipMemcpy(states_raw, d_states_, nr_poses_ * n_features_ * sizeof(float), hipMemcpyDeviceToHost);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpy d_states -> states");
    #endif


    for (int i = 0; i < nr_poses_; i++) {
        for (int j = 0; j < n_features_; j++) {
            states[i][j] = states_raw[i * n_features_ + j];
        }
    }
}




void CudaFilter::propagate_multiple(const float &current_time, vector<vector<vector<float> > > &states)
{

    float delta_time = current_time - last_propagation_time_;
    last_propagation_time_ = current_time;

    int n_objects = states[0].size();

    float *states_raw = (float*) malloc(nr_poses_ * n_objects * n_features_ * sizeof(float));
    for (int i = 0; i < nr_poses_; i++) {
        for (int j = 0; j < n_objects; j++) {
            for (int k = 0; k < n_features_; k++) {
                states_raw[(i * n_objects *n_features_) + j * n_features_ + k] = states[i][j][k];
            }
        }
    }


    hipMemcpy(d_states_, states_raw, nr_poses_ * n_objects * n_features_ * sizeof(float), hipMemcpyHostToDevice);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpy states -> d_states");
    #endif


    propagate_multiple_kernel <<< n_blocks_, nr_threads_ >>> (d_states_, nr_poses_, n_objects, n_features_, delta_time, d_mrg_states_);
    #ifdef CHECK_ERRORS
        check_cuda_error("propagate kernel call");
    #endif



    // TODO necessary? Remove for performance?
    hipDeviceSynchronize();
    #ifdef CHECK_ERRORS
        check_cuda_error("hipDeviceSynchronize propagate");
    #endif



    hipMemcpy(states_raw, d_states_, nr_poses_ * n_objects * n_features_ * sizeof(float), hipMemcpyDeviceToHost);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpy d_states -> states");
    #endif


    for (int i = 0; i < nr_poses_; i++) {
        for (int j = 0; j < n_objects; j++) {
            for (int k = 0; k < n_features_; k++) {
                states[i][j][k] = states_raw[(i * n_objects *n_features_) + j * n_features_ + k];
            }
        }
    }

    free(states_raw);
}




void CudaFilter::compare(float observation_time, bool constant_occlusion, vector<float> &log_likelihoods) {

#ifdef PROFILING_ACTIVE
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;
#endif

    dim3 gridDim = dim3(nr_poses_per_row_, nr_poses_per_column_);

    // update observation time
    float delta_time = observation_time - occlusion_time_;
    occlusion_time_ = observation_time;



#ifdef PROFILING_ACTIVE
    hipEventRecord(start);
#endif

    compare_kernel <<< gridDim, 128 >>> (d_observations_, d_visibility_probs_, n_cols_ * n_rows_,
            constant_occlusion, d_log_likelihoods_, delta_time, nr_poses_, n_rows_, n_cols_);
    #ifdef CHECK_ERRORS
        check_cuda_error("compare kernel call");
    #endif

    hipDeviceSynchronize();
    #ifdef CHECK_ERRORS
        check_cuda_error("hipDeviceSynchronize compare");
    #endif

#ifdef PROFILING_ACTIVE
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);    
    count_++;
    compare_kernel_time_ += milliseconds;
    if (count_ == COUNT) {
        cout << "compare kernel: " << compare_kernel_time_ * 1e3 / count_ << " us" << endl;
    }
    hipEventRecord(start);
#endif

    hipMemcpy(&log_likelihoods[0], d_log_likelihoods_, nr_poses_ * sizeof(float), hipMemcpyDeviceToHost);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpy d_log_likelihoods -> log_likelihoods");
    #endif

    hipDeviceSynchronize();
    #ifdef CHECK_ERRORS
        check_cuda_error("hipDeviceSynchronize compare");
    #endif

#ifdef PROFILING_ACTIVE
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    copy_likelihoods_time_ += milliseconds;
    if (count_ == COUNT) {
        cout << "copy likelihoods: " << copy_likelihoods_time_ * 1e3 / count_ << " us" << endl;
    }
#endif
}



void CudaFilter::compare_multiple(bool update, vector<float> &log_likelihoods) {

#ifdef PROFILING_ACTIVE
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;
    hipEventRecord(start);
#endif

    double delta_time = observation_time_ - occlusion_time_;
    if(update) occlusion_time_ = observation_time_;
//    cout << "delta time: " << delta_time << endl;


    compare_multiple_kernel <<< grid_dimension_, nr_threads_ >>> (d_observations_, d_visibility_probs_, d_visibility_probs_copy_, d_prev_sample_indices_, n_cols_ * n_rows_,
                                           d_log_likelihoods_, delta_time, nr_poses_, n_rows_, n_cols_, update);
    #ifdef CHECK_ERRORS
        check_cuda_error("compare kernel call");
    #endif

    hipDeviceSynchronize();
    #ifdef CHECK_ERRORS
        check_cuda_error("hipDeviceSynchronize compare_multiple");
    #endif



    // switch to new / copied visibility probabilities
    if (update) {
        float *tmp_pointer;
        tmp_pointer = d_visibility_probs_;
        d_visibility_probs_ = d_visibility_probs_copy_;
        d_visibility_probs_copy_ = tmp_pointer;
    }

#ifdef PROFILING_ACTIVE
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    count_++;
    compare_kernel_time_ += milliseconds;
    if (count_ == COUNT) {
        cout << "compare kernel: " << compare_kernel_time_ * 1e3 / count_ << " us" << endl;
    }
    hipEventRecord(start);
#endif

    hipMemcpy(&log_likelihoods[0], d_log_likelihoods_, nr_poses_ * sizeof(float), hipMemcpyDeviceToHost);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpy d_log_likelihoods -> log_likelihoods");
    #endif

    hipDeviceSynchronize();
    #ifdef CHECK_ERRORS
        check_cuda_error("hipDeviceSynchronize compare");
    #endif

#ifdef PROFILING_ACTIVE
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    copy_likelihoods_time_ += milliseconds;
    if (count_ == COUNT) {
        cout << "copy likelihoods: " << copy_likelihoods_time_ * 1e3 / count_ << " us" << endl;
    }
#endif
}







void CudaFilter::resample(vector<int> resampling_indices) {

//    cout << "resample <<< " << n_poses_ << ", " << 128 << " >>>" << endl;

    hipMemcpy(d_resampling_indices_, &resampling_indices[0], sizeof(int) * nr_poses_, hipMemcpyHostToDevice);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpy resampling_indices -> d_resampling_indices_");
    #endif

//        int min = 100;
//        int max = -1;
//        for (int i = 0; i < resampling_indices.size(); i++) {
//            int value = resampling_indices[i];
//            if (value > max) max = value;
//            if (value < min) min = value;
//        }
//        cout << "resample min: " << min << ", max: " << max << endl;


    int nr_pixels = n_rows_ * n_cols_;

    resample_kernel <<< nr_poses_, 128 >>> (d_visibility_probs_, d_visibility_probs_copy_,
                                    d_states_, d_states_copy_,
                                    d_resampling_indices_, nr_pixels, n_features_);
    #ifdef CHECK_ERRORS
        check_cuda_error("resample kernel call");
    #endif

    hipDeviceSynchronize();
    #ifdef CHECK_ERRORS
        check_cuda_error("hipDeviceSynchronize resample");
    #endif


    // switch the visibility probs pointers, so that the next Compare() call will access the resampled
    // visibility probs. Same for the states.
    float *tmp_pointer;
    tmp_pointer = d_visibility_probs_;
    d_visibility_probs_ = d_visibility_probs_copy_;
    d_visibility_probs_copy_ = tmp_pointer;
    tmp_pointer = d_states_;
    d_states_ = d_states_copy_;
    d_states_copy_ = tmp_pointer;

}




void CudaFilter::resample_multiple(vector<int> resampling_indices) {

    hipMemcpy(d_resampling_indices_, &resampling_indices[0], sizeof(int) * nr_poses_, hipMemcpyHostToDevice);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpy resampling_indices -> d_resampling_indices_");
    #endif

    int nr_pixels = n_rows_ * n_cols_;

    resample_multiple_kernel <<< nr_poses_, 128 >>> (d_visibility_probs_, d_visibility_probs_copy_,
                                             d_resampling_indices_, nr_pixels);
    #ifdef CHECK_ERRORS
        check_cuda_error("resample kernel call");
    #endif

    hipDeviceSynchronize();
    #ifdef CHECK_ERRORS
        check_cuda_error("hipDeviceSynchronize resample");
    #endif


    // switch the visibility probs pointers, so that the next Compare() call will access the resampled
    // visibility probs.
    float *tmp_pointer;
    tmp_pointer = d_visibility_probs_;
    d_visibility_probs_ = d_visibility_probs_copy_;
    d_visibility_probs_copy_ = tmp_pointer;
}





// ===================================================================================== //
// =============================== CUDA FILTER SETTERS ================================= //
// ===================================================================================== //

void CudaFilter::set_states(std::vector<std::vector<float> > &states, int seed)
{
    if (n_poses_set_) {
        // copy states into linear array
        /* TODO maybe padding can speed up the memory accesses later from the kernel, since
         * right now, each MP needs 7 values out of d_states_. 8 would be a much better number. */
        n_features_ = states[0].size();

        int states_size = nr_poses_ * n_features_ * sizeof(float);
        float *states_raw = (float*) malloc(states_size);

        for (size_t i = 0; i < nr_poses_; i++) {
            for (size_t j = 0; j < n_features_; j++) {
                states_raw[i * n_features_ + j] = states[i][j];
            }
        }

        allocate(d_states_, states_size, "d_states");
        allocate(d_states_copy_, states_size, "d_states_copy");     // placeholder for resampling purposes

        hipMemcpy(d_states_, states_raw, states_size, hipMemcpyHostToDevice);
        #ifdef CHECK_ERRORS
            check_cuda_error("hipMemcpy states_raw -> d_states_");
        #endif

        free(states_raw);

        // setup random number generators for each thread to be used in the propagate kernel
        allocate(d_mrg_states_, nr_poses_ * sizeof(hiprandStateMRG32k3a), "d_mrg_states");

        setup_number_generators_kernel <<< n_blocks_, nr_threads_ >>> (seed, d_mrg_states_, nr_poses_);

        hipDeviceSynchronize();
    } else {
        cout << "WARNING: set_states() was not executed, because n_poses_ has not been set previously";
        exit(-1);
    }
}






void CudaFilter::set_states_multiple(int n_objects, int n_features, int seed)
{
    if (n_poses_set_) {
        n_features_ = n_features;

        int states_size = nr_poses_ * n_objects * n_features_ * sizeof(float);
        allocate(d_states_, states_size, "d_states");


        // setup random number generators for each thread to be used in the propagate kernel
        allocate(d_mrg_states_, nr_poses_ * sizeof(hiprandStateMRG32k3a), "d_mrg_states");

        setup_number_generators_kernel <<< n_blocks_, nr_threads_ >>> (seed, d_mrg_states_, nr_poses_);

        hipDeviceSynchronize();
    } else {
        cout << "WARNING: set_states_multiple() was not executed, because n_poses_ has not been set previously";
        exit(-1);
    }
}







void CudaFilter::set_observations(const float* observations, const float observation_time) {

//    delta_time_ = observation_time - last_observation_time_;
    observation_time_ = observation_time;
//    cout << "delta time: " << delta_time_ << ", last_observation_time: " << occlusion_time_ << endl;
    set_observations(observations);
}

void CudaFilter::set_observations(const float* observations) {
    hipMemcpy(d_observations_, observations, n_cols_ * n_rows_ * sizeof(float), hipMemcpyHostToDevice);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpy observations -> d_observations_");
    #endif
    hipDeviceSynchronize();
}


void CudaFilter::set_prev_sample_indices(const int* prev_sample_indices) {
    hipMemcpy(d_prev_sample_indices_, prev_sample_indices, nr_poses_ * sizeof(int), hipMemcpyHostToDevice);
//    cout << "when setting prev_sample_indices: n_poses: " << n_poses_ << ", max poses: " << n_max_poses_ << endl;
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpy prev_sample_indices -> d_prev_sample_indices_");
    #endif
    hipDeviceSynchronize();
}


void CudaFilter::set_resolution(const int n_rows, const int n_cols, int& nr_poses, int& nr_poses_per_row, int& nr_poses_per_column) {
    n_rows_ = n_rows;
    n_cols_ = n_cols;

    // reallocate buffers
    allocate(d_observations_, n_cols_ * n_rows_ * sizeof(float), "d_observations");    
    allocate_memory_for_max_poses(nr_poses, nr_poses_per_row, nr_poses_per_column);
}


void CudaFilter::set_visibility_probabilities(const float* visibility_probabilities) {
    hipMemcpy(d_visibility_probs_, visibility_probabilities, n_rows_ * n_cols_ * nr_poses_ * sizeof(float), hipMemcpyHostToDevice);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpy visibility_probabilities -> d_visibility_probs_");
    #endif
}




void CudaFilter::allocate_memory_for_max_poses(int& allocated_poses,
                                               int& allocated_poses_per_row,
                                               int& allocated_poses_per_column) {

    // check limitation by global memory
    size_t size_of_log_likelihoods = sizeof(float) * allocated_poses;
    size_t size_of_resampling_indices = sizeof(int) * allocated_poses;
    size_t size_of_prev_sample_indices = sizeof(int) * allocated_poses;
    size_t size_of_visibility_probs = n_rows_ * n_cols_ * allocated_poses * sizeof(float);
    size_t size_of_opengl_textures = size_of_visibility_probs * 2;
    size_t size_of_observations = n_cols_ * n_rows_ * sizeof(float);

    size_t total_size = size_of_log_likelihoods + size_of_resampling_indices + size_of_prev_sample_indices
                      + size_of_visibility_probs * 2 + size_of_opengl_textures + size_of_observations;

    if (total_size > cuda_device_properties_.totalGlobalMem) {

        std::cout << "The space (" << total_size << " B) for the number of maximum poses you requested (" << allocated_poses << ") cannot be allocated. "
                  << "The limit is global memory size (" << cuda_device_properties_.totalGlobalMem
                  << " B) retrieved from CUDA properties." << std::endl;

        size_t size_depending_on_nr_poses = (sizeof(float) + sizeof(int) * 2 + n_rows_ * n_cols_ * sizeof(float) * 4);
        allocated_poses = min(allocated_poses, (int) floor((cuda_device_properties_.totalGlobalMem - size_of_observations) / size_depending_on_nr_poses));
        allocated_poses_per_column = ceil(allocated_poses / allocated_poses_per_row);

        std::cout << "Instead, space (" << allocated_poses * size_depending_on_nr_poses + size_of_observations << " B) for " << allocated_poses << " poses was allocated. " << std::endl;
    }


    // check limitation by texture size
    if (cuda_device_properties_.maxTexture2D[0] <= allocated_poses_per_row * n_cols_) {

        std::cout << "The max poses you requested (" << allocated_poses << ") could not be allocated." << std::endl;

        allocated_poses_per_row = cuda_device_properties_.maxTexture2D[0] / n_cols_;
        allocated_poses_per_column = ceil(allocated_poses / allocated_poses_per_row);

        if (cuda_device_properties_.maxTexture2D[1] <= allocated_poses_per_column * n_rows_) {
            allocated_poses_per_column = cuda_device_properties_.maxTexture2D[1] / n_rows_;
        }

        allocated_poses = min(allocated_poses, allocated_poses_per_row * allocated_poses_per_column);

        std::cout << "The limit is max texture size (" << cuda_device_properties_.maxTexture2D[0]
                  << ", " << cuda_device_properties_.maxTexture2D[1] << ") retrieved from CUDA properties. "
                  << "Number of poses was reduced to (" << allocated_poses_per_row << ", "
                  << allocated_poses_per_column << "), a total of " << allocated_poses << std::endl;
    }

    nr_max_poses_ = allocated_poses;
    nr_max_poses_per_row_ = allocated_poses_per_row;
    nr_max_poses_per_column_ = allocated_poses_per_column;


/*
    nr_max_poses_ = n_poses;
    nr_max_poses_per_row_ = n_poses_x;

    // determine n_max_poses_y_
    nr_max_poses_per_column_ = nr_max_poses_ / nr_max_poses_per_row_;
    if (n_poses % nr_max_poses_per_row_ != 0) nr_max_poses_per_column_++;

    n_poses_ = nr_max_poses_;
    n_poses_x_ = nr_max_poses_per_row_;
    n_poses_y_ = nr_max_poses_per_column_;

*/
    n_poses_set_ = true;

    bool nr_poses_changed = false;
    set_default_kernel_config(nr_max_poses_, nr_max_poses_per_row_, nr_max_poses_per_column_, nr_poses_changed);

    allocated_poses = nr_max_poses_;
    allocated_poses_per_row = nr_max_poses_per_row_;
    allocated_poses_per_column = nr_max_poses_per_column_;

    nr_poses_ = nr_max_poses_;
    nr_poses_per_row_ = nr_max_poses_per_row_;
    nr_poses_per_column_ = nr_max_poses_per_column_;

    if (nr_poses_changed) {
        size_of_log_likelihoods = sizeof(float) * nr_max_poses_;
        size_of_resampling_indices = sizeof(int) * nr_max_poses_;
        size_of_prev_sample_indices = sizeof(int) * nr_max_poses_;
        size_of_visibility_probs = n_rows_ * n_cols_ * nr_max_poses_ * sizeof(float);
    }


    // reallocate arrays
    allocate(d_log_likelihoods_, size_of_log_likelihoods, "d_log_likelihoods");
    allocate(d_resampling_indices_, size_of_resampling_indices, "d_resampling_indices");
    allocate(d_prev_sample_indices_, size_of_prev_sample_indices, "d_prev_sample_indices");
    allocate(d_visibility_probs_, size_of_visibility_probs, "d_visibility_probs");
    allocate(d_visibility_probs_copy_, size_of_visibility_probs, "d_visibility_probs_copy");

    // TODO maybe delete after set_visibility_probabilities is properly in use
    vector<float> initial_visibility_probs (n_rows_ * n_cols_ * nr_max_poses_, visibility_prob_default_);
    hipMemcpy(d_visibility_probs_, &initial_visibility_probs[0], size_of_visibility_probs, hipMemcpyHostToDevice);
    #ifdef CHECK_ERRORS
        check_cuda_error("hipMemcpy visibility_prob_default_ -> d_visibility_probs_");
    #endif


    hipDeviceSynchronize();
    #ifdef CHECK_ERRORS
        check_cuda_error("hipDeviceSynchronize allocate_memory_for_max_poses");
    #endif


}


void CudaFilter::set_number_of_poses(int& nr_poses, int& nr_poses_per_row, int& nr_poses_per_column) {
    if (nr_poses <= nr_max_poses_) {

        if (nr_max_poses_per_row_ < nr_poses_per_row) {
            nr_poses_per_row = nr_max_poses_per_row_;
            nr_poses_per_column = ceil(nr_poses / nr_poses_per_row);
            if (nr_max_poses_per_column_ < nr_poses_per_column) {
                nr_poses_per_column = nr_max_poses_per_column_;
            }

            std::cout << "Number of poses was reduced to (" << nr_poses_per_row << ", "
                      << nr_poses_per_column << ") because of the maximum number of poses set in the beginning." << std::endl;
        }

        nr_poses = min(nr_poses, nr_poses_per_row * nr_poses_per_column);

        nr_poses_ = nr_poses;
        nr_poses_per_row_ = nr_poses_per_row;
        nr_poses_per_column_ = nr_poses_per_column;

        /*


        nr_poses_ = nr_poses;
        nr_poses_per_row_ = nr_poses_per_row;

        // determine n_max_poses_y_
        nr_poses_per_column_ = nr_poses_ / nr_poses_per_row_;
        if (nr_poses % nr_poses_per_row_ != 0) nr_poses_per_column_++;

        if (nr_poses_per_row_ > nr_max_poses_per_row_ || nr_poses_per_column_ > nr_max_poses_per_column_) {
            cout << "WARNING: You tried to evaluate more poses in a row or in a column than was allocated in the beginning."
                 << endl << "Check set_number_of_poses() functions in object_rasterizer.cpp" << endl;
        }*/

        bool nr_poses_changed = false;
        set_default_kernel_config(nr_poses_, nr_poses_per_row_, nr_poses_per_column_, nr_poses_changed);

        nr_poses = nr_poses_;
        nr_poses_per_row = nr_poses_per_row_;
        nr_poses_per_column = nr_poses_per_column_;

    } else {
        cout << "ERROR (Cuda): You tried to evaluate more poses (" << nr_poses << ") than specified by max_poses (" << nr_max_poses_ << ")" << endl;
        exit(-1);
    }
}



void CudaFilter::set_default_kernel_config(int& nr_poses, int& nr_poses_per_row, int& nr_poses_per_column,
                                           bool& nr_poses_changed) {
    nr_threads_ = min(DEFAULT_NR_THREADS, cuda_device_properties_.maxThreadsDim[0]);

    // check for grid dimension limitations
    if (cuda_device_properties_.maxGridSize[0] < nr_poses_per_row) {
        nr_poses_per_row = cuda_device_properties_.maxGridSize[0];
        nr_poses_per_column = ceil(nr_poses / nr_poses_per_row);
        if (cuda_device_properties_.maxGridSize[1] < nr_poses_per_column) {
            nr_poses_per_column = cuda_device_properties_.maxGridSize[1];
        }

        nr_poses = min(nr_poses, nr_poses_per_row * nr_poses_per_column);

        nr_poses_changed = true;

        std::cout << "Number of poses was reduced to (" << nr_poses_per_row << ", "
                  << nr_poses_per_column << ") because of the maximum grid size ("
                  << cuda_device_properties_.maxGridSize[0] << ", " << cuda_device_properties_.maxGridSize[1]
                  << ") retrieved from CUDA properties." << std::endl;
    }


    grid_dimension_ = dim3(nr_poses_per_row, nr_poses_per_column);


    /*

    // determine n_threads_ and n_blocks_
    // n_threads_ should lie between 32 (warp_size) and 128 and all microprocessors should be busy
    nr_threads_ = ((nr_poses_ / n_mps_) / warp_size_) * warp_size_;
    if (nr_threads_ == 0) nr_threads_ = warp_size_;
    if (nr_threads_ > 4 * warp_size_) nr_threads_ = 4 * warp_size_;

    n_blocks_ = nr_poses_ / nr_threads_;
    if (n_blocks_ % nr_poses_ != 0) n_blocks_++;

    */
}


void CudaFilter::set_texture_array(hipArray_t texture_array) {
    d_texture_array_ = texture_array;
}


// ===================================================================================== //
// =============================== CUDA FILTER GETTERS ================================= //
// ===================================================================================== //




vector<float> CudaFilter::get_visibility_probabilities(int state_id) {
//    cout << "n_rows_: " << n_rows_ << ", n_cols_: " << n_cols_ << endl;
    float* visibility_probabilities = (float*) malloc(n_rows_ * n_cols_ * sizeof(float));
    int offset = state_id * n_rows_ * n_cols_;
    hipMemcpy(visibility_probabilities, d_visibility_probs_ + offset, n_rows_ * n_cols_ * sizeof(float), hipMemcpyDeviceToHost);
#ifdef CHECK_ERRORS
    check_cuda_error("hipMemcpy d_visibility_probabilities -> visibility_probabilities");
#endif
    vector<float> visibility_probabilities_vector;
    for (int i = 0; i < n_rows_ * n_cols_; i++) {
        visibility_probabilities_vector.push_back(visibility_probabilities[i]);
    }
    free(visibility_probabilities);
    return visibility_probabilities_vector;
}



vector<vector<float> > CudaFilter::get_visibility_probabilities() {
    float* visibility_probabilities = (float*) malloc(nr_poses_ * n_rows_ * n_cols_ * sizeof(float));
    hipMemcpy(visibility_probabilities, d_visibility_probs_, nr_poses_ * n_rows_ * n_cols_ * sizeof(float), hipMemcpyDeviceToHost);
#ifdef CHECK_ERRORS
    check_cuda_error("hipMemcpy d_visibility_probabilities -> visibility_probabilities");
#endif
    vector<vector<float> > visibility_probabilities_vector;
    vector<float> tmp_vector (n_rows_ * n_cols_);
    for (int i = 0; i < nr_poses_; i++) {
        for (int j = 0; j < n_rows_ * n_cols_; j++) {
            tmp_vector[j] = visibility_probabilities[i * n_rows_ * n_cols_ + j];
        }
        visibility_probabilities_vector.push_back(tmp_vector);
    }
    return visibility_probabilities_vector;
}



// ===================================================================================== //
// ========================== CUDA FILTER HELPER FUNCTIONS ============================= //
// ===================================================================================== //




template <typename T> void CudaFilter::allocate(T * &pointer, size_t size, string name) {
#ifdef CHECK_ERRORS
    size_t free_space_before, free_space_after, total_space;
    hipMemGetInfo(&free_space_before, &total_space);
#endif
    hipFree(pointer);
    hipMalloc((void **) &pointer, size);
#ifdef CHECK_ERRORS
    hipMemGetInfo(&free_space_after, &total_space);
    std::cout << "memory to allocate for " << name << ": " << size / 1e6 << " MB; free space: " << free_space_before / 1e6
         << "MB; --> allocated " << (free_space_before - free_space_after) / 1e6 << " MB, free space left: " << free_space_after / 1e6 << " MB" << std::endl;
    check_cuda_error("hipMalloc failed");
#endif
}



void CudaFilter::map_texture() {
    hipBindTextureToArray(texture_reference, d_texture_array_);
    check_cuda_error("hipBindTextureToArray");
}




void CudaFilter::check_cuda_error(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

CudaFilter::~CudaFilter() {
    hipFree(d_states_);
    hipFree(d_states_copy_);
    hipFree(d_visibility_probs_);
    hipFree(d_visibility_probs_copy_);
    hipFree(d_observations_);
    hipFree(d_log_likelihoods_);
    hipFree(d_mrg_states_);
    hipFree(d_resampling_indices_);

}

void CudaFilter::destroy_context() {
    hipFree(d_states_);
    hipFree(d_states_copy_);
    hipFree(d_visibility_probs_);
    hipFree(d_visibility_probs_copy_);
    hipFree(d_observations_);
    hipFree(d_log_likelihoods_);
    hipFree(d_mrg_states_);
    hipFree(d_resampling_indices_);
    hipDeviceReset();
}

}

